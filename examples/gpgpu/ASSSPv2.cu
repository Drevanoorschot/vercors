/***********************************************************************************
Created by Mohsen Safari.
************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>


//@ pure int inf() = -1; //100000000; any negative integer

/*@ 
pure bool Graph(int V, int A, seq<int> start, seq<int> end, seq<int> weight) =
	0 < V &&
	0 < A && |start| == A && |end| == A && |weight| == A &&
	(\forall int i; 0 <= i && i < A; 
		0 <= start[i] && start[i] < V && 
		0 <= end[i] && end[i] < V && 
		start[i] != end[i] && 
		(\forall int j; 0 <= j && j < A && i != j && start[i] == start[j]; end[i] != end[j]) &&
     weight[i] > 0); 
@*/

/*@ 
requires Graph(V, A, start, end, weight);
ensures (\forall int i; i >= 0 && i < A; Path(V, A, start, end, weight, start[i], end[i], seq<int> { i }));
pure bool Path(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) =
	0 <= x && x < V &&
	0 <= y && y < V &&
	(\forall int i; 0 <= i && i < |P|; 0 <= P[i] && P[i] < A) &&
	(0 == |P| ==> x == y) &&
	(0 < |P| ==> start[P[0]] == x && end[P[|P| - 1]] == y) &&
	(\forall int i; 0 <= i && i < |P| - 1; end[P[i]] == start[P[i + 1]]);
@*/

/*@ 
requires Graph(V, A, start, end, weight);
ensures (\forall int i; i >= 0 && i < A; SimplePath(V, A, start, end, weight, start[i], end[i], seq<int> { i }));
pure bool SimplePath(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) =
  Path(V, A, start, end, weight, x, y, P) &&
  (\forall int i; 0 <= i && i < |P| - 1; (\forall int j; i < j && j < |P|; start[P[i]] != end[P[j]])); 
@*/

/*@ 
requires Graph(V, A, start, end, weight);
pure bool ExPath(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, int len) =
	(\exists seq<int> P; Path(V, A, start, end, weight, x, y, P); |P| <= len);
@*/

/*@   
requires Graph(V, A, start, end, weight);
ensures (\forall int i; i >= 0 && i < A; ExPathEqual(V, A, start, end, weight, start[i], end[i], weight[i]));
ensures ExPathEqual(V, A, start, end, weight, x, x, 0);
pure bool ExPathEqual(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, int len) =
	(\exists seq<int> P; Path(V, A, start, end, weight, x, y, P); cost(V, A, start, end, weight, P) == len);
@*/

/*@
requires Graph(V, A, start, end, weight);
requires (\forall int i; 0 <= i && i < |P|; 0 <= P[i] && P[i] < A);
ensures 0 <= \result;
ensures 0 == |P| ==> \result == 0;
ensures (P != seq<int>{}) ==> \result > 0;
ensures (\result == 0) ==> (P == seq<int>{});
ensures (\result != 0) ==> (P != seq<int>{});
ensures 1 == |P| ==> \result == weight[P[0]];
pure int cost(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> P) =  
  0 < |P| ? weight[P[0]] + cost(V, A, start, end, weight, tail(P)) : 0;
@*/  

/*@
ensures (xs + ys) + zs == xs + (ys + zs);
pure bool iseq_assoc(seq<int> xs, seq<int> ys, seq<int> zs) = true;
@*/

/*@
requires Graph(V, A, start, end, weight);
requires Path(V, A, start, end, weight, x, y, P);
requires 0 <= a && a < A;
requires end[a] == x;
ensures \result && Path(V, A, start, end, weight, start[a], y, seq<int> { a } + P);
pure bool lemma_path_append(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P, int a);
@*/

/*@
requires Graph(V, A, start, end, weight);
requires Path(V, A, start, end, weight, s, t, P);
requires Path(V, A, start, end, weight, t, u, Q);
ensures \result && Path(V, A, start, end, weight, s, u, P + Q);
pure bool lemma_path_trans(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int s, int t, int u, seq<int> P, seq<int> Q) =
	0 < |P| ==> (
		lemma_path_trans(V, A, start, end, weight, end[P[0]], t, u, tail(P), Q) &&
		lemma_path_append(V, A, start, end, weight, end[P[0]], u, tail(P) + Q, P[0]) &&
		iseq_assoc(seq<int> { P[0] }, tail(P), Q) &&
		seq<int> { P[0] } + tail(P) == P
	); 
@*/

/*@
requires 0 < |xs|;
ensures (xs + ys)[0] == xs[0];
ensures tail(xs + ys) == tail(xs) + ys;
ensures \result;  
pure bool lemma_app_nonempty(seq<int> xs, seq<int> ys) = true; 
@*/

/*@
requires 0 < |ys|;
ensures \result && ((xs + seq<int>{ys[0]}) + tail(ys)) == xs + ys;
pure bool lemma_app_append_right(seq<int> xs, seq<int> ys) =
  iseq_assoc(xs, seq<int>{ys[0]}, tail(ys));
@*/

/*@
requires Graph(V, A, start, end, weight);
requires (\forall int i; 0 <= i && i < |P|; 0 <= P[i] && P[i] < A);
requires (\forall int i; 0 <= i && i < |Q|; 0 <= Q[i] && Q[i] < A);
ensures \result && cost(V, A, start, end, weight, P + Q) == cost(V, A, start, end, weight, P) + cost(V, A, start, end, weight, Q);
pure bool lemma_cost_app(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> P, seq<int> Q) =  
  0 < |P| ==> lemma_app_nonempty(P, Q) && lemma_cost_app(V, A, start, end, weight, tail(P), Q);
@*/

/*@
requires Graph(V, A, start, end, weight);
requires Path(V, A, start, end, weight, s, t, P);
requires Path(V, A, start, end, weight, t, u, Q);
ensures \result && Path(V, A, start, end, weight, s, u, P + Q);
ensures \result && cost(V, A, start, end, weight, P + Q) == cost(V, A, start, end, weight, P) + cost(V, A, start, end, weight, Q);
pure bool lemma_cost_path(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int s, int t, int u, seq<int> P, seq<int> Q) =
  lemma_path_trans(V, A, start, end, weight, s, t, u, P, Q) &&
  lemma_cost_app(V, A, start, end, weight, P, Q);
@*/

/*@
requires b;
ensures \result == xs;
pure seq<int> assertSeq(seq<int> xs, bool b) = xs;
@*/

/*@
requires Graph(V, A, start, end, weight);
ensures \result && (\forall int i; i >= 0 && i < A; Path(V, A, start, end, weight, start[i], end[i], seq<int>{i}));
ensures	\result && (\forall int i; i >= 0 && i < A; ExPathEqual(V, A, start, end, weight, start[i], end[i], weight[i]));	
pure bool lemma_one_path(int V, int A, seq<int> start, seq<int> end, seq<int> weight) = true;
@*/


/* ** Invariant 10 */
/*@
requires Graph(V, A, start, end, weight);
requires ExPathEqual(V, A, start, end, weight, s, t, len1); 
requires ExPathEqual(V, A, start, end, weight, t, u, len2); 
ensures \result && ExPathEqual(V, A, start, end, weight, s, u, len1+len2);
pure bool lemma_expath_trans(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int s, int t, int u, int len1, int len2) =
	(\forall seq<int> P; Path(V, A, start, end, weight, s, t, P);
		(\forall seq<int> Q; Path(V, A, start, end, weight, t, u, Q);
			lemma_cost_path(V, A, start, end, weight, s, t, u, P, Q)));
@*/


/*@
  requires i >= 0 && i <= |xs|-1;
  ensures |\result| == |xs| - 1 - i;
  ensures (\forall int k; k >= 0 && k < |\result|; \result[k] == xs[i+k]);
  ensures i == 0 ==> (\forall int k; k >= 0 && k < |xs|-1; \result[k] == xs[k]);
  pure seq<int> subseq(seq<int> xs, int i) = (i != |xs|-1) ? seq<int>{xs[i]} + subseq(xs, i+1) : seq<int>{}; 
@*/







////////////////////////////////////////////////////////////////////////////////
//compare
////////////////////////////////////////////////////////////////////////////////  
  /*@
  requires start_cost == inf() || start_cost >= 0;
  requires end_cost == inf() || end_cost >= 0;
  requires weight > 0;
  ensures (start_cost != inf()) && (end_cost == inf() || start_cost + weight <= end_cost) ==> \result;
  ensures \result ==> (start_cost != inf()) && (end_cost == inf() || start_cost + weight <= end_cost);
  ensures (start_cost == inf()) || (end_cost != inf() && start_cost + weight > end_cost) ==> !\result;
  ensures !\result ==> (start_cost == inf()) || (end_cost != inf() && start_cost + weight > end_cost);
  ensures (\result && end_cost != inf()) ==> start_cost + weight <= end_cost;
  pure bool compare(int start_cost, int weight, int end_cost) = 
    (start_cost != inf() && (end_cost == inf() || start_cost+weight <= end_cost)) ? true : false;
  @*/


////////////////////////////////////////////////////////////////////////////////
//@ ensures \pointer(\result, N, write);
int *vercorsMallocInt(int N);
void vercorsFreeInt(int *ar);
//@ ensures \pointer(\result, N, write);
int *vercorsCudaMallocInt(int N);
void vercorsCudaFreeInt(int *addr);
void vercorsCudaMemcpyInt(int *tgt, int *src, int N, int direction); 

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) 
{
	
	int V = 1024; // no. of vertices
	int A = 10 * V; // no. of arcs
    	
	// allocate host memory
  int* host_start = vercorsMallocInt(A);
  int* host_end = vercorsMallocInt(A);
  int* host_weight = vercorsMallocInt(A);
  int* host_cost = vercorsMallocInt(V);
  
  int source = V/2;
  
  // these assumptions come from the input graph file which we change it into assertions 
  //@ assume (\forall int i; 0 <= i && i < A; host_start[i] >= 0 && host_start[i] < V);
  //@ assume (\forall int i; 0 <= i && i < A; host_end[i] >= 0 && host_end[i] < V);
  //@ assume (\forall int i; 0 <= i && i < A; host_start[i] != host_end[i]); // No self-loop
  
  //@ assume (\forall int i; 0 <= i && i < A; host_weight[i] > 0);
  
  //@ assume (\forall int i; 0 <= i && i < A; (\forall int j; 0 <= j && j < A && i != j; host_start[i] == host_start[j] ==> host_end[i] != host_end[j])); 
  
  //@ loop_invariant i >= 0 && i <= V;
  //@ loop_invariant (\forall* int k; 0 <= k && k < V; Perm(host_cost[k], write));
  //@ loop_invariant (\forall int k; 0 <= k && k < i; host_cost[k] == -1);   
  for(unsigned int i = 0; i < V; i++) 
  {
    host_cost[i] = -1;
  }
  host_cost[source] = 0;
  
  //@ assert (\forall int i; 0 <= i && i < V && i != source; host_cost[i] == inf());
  
  //@ assert host_cost[source] == 0;

  //Copy the arrays to device memory
  int* device_start;
  device_start = vercorsCudaMallocInt(A);
  vercorsCudaMemcpyInt( device_start, host_start, A, hipMemcpyHostToDevice) ;
  //@ assume (\forall int i; i >= 0 && i < A; host_start[i] == device_start[i]);

  //Copy the arrays to device memory
  int* device_end;
  device_end = vercorsCudaMallocInt(A);
  vercorsCudaMemcpyInt( device_end, host_end, A, hipMemcpyHostToDevice) ;
  //@ assume (\forall int i; i >= 0 && i < A; host_end[i] == device_end[i]);
  
  //Copy the arrays to device memory
  int* device_weight;
  device_weight = vercorsCudaMallocInt(A);
  vercorsCudaMemcpyInt( device_weight, host_weight, A, hipMemcpyHostToDevice) ;
  //@ assume (\forall int i; i >= 0 && i < A; host_weight[i] == device_weight[i]);


  //Copy the arrays to device memory
  int* device_cost;
  device_cost = vercorsCudaMallocInt(V);
  vercorsCudaMemcpyInt( device_cost, host_cost, V, hipMemcpyHostToDevice) ;
  //@ assume (\forall int i; i >= 0 && i < V; host_cost[i] == device_cost[i]);
  
    
  //setup execution parameters
	int num_of_blocks = 10;
	int num_of_threads_per_block = V;
  
  //dim3  grid( num_of_blocks, 1, 1); //grid has three parameters to indicate the dimensions. Here we have one dimensional grid (of blocks). It can be one, two or three dimensions.
  //dim3  threads( num_of_threads_per_block, 1, 1); //threads indicates the dimensions of one block. Here each block has one dimension (of threads). It can be one, two or three dimensions.
  
  int counter = 0;

  //@ ghost seq<int> contrib; 
  //@ assume |contrib| == A && (\forall int i; i >= 0 && i < A; contrib[i] == 0);
  
  //@ ghost seq<int> contrib2; 
  //@ assume |contrib2| == A && (\forall int i; i >= 0 && i < A; contrib2[i] == 0);
  
  //@ ghost seq<int> cost_seq ;
  //@ assume |cost_seq| == V && (\forall int i; i >= 0 && i < V; cost_seq[i] == host_cost[i]);
  
  //@ ghost seq<int> oldcost_seq ; 
  //@ assume |oldcost_seq| == V && (\forall int i; i >= 0 && i < V; oldcost_seq[i] == host_cost[i]);
  
  //@ ghost seq<int> start_seq ; 
  //@ assume |start_seq| == A && (\forall int i; i >= 0 && i < A; start_seq[i] == host_start[i]);
  
  //@ ghost seq<int> end_seq ;
  //@ assume |end_seq| == A && (\forall int i; i >= 0 && i < A; end_seq[i] == host_end[i]);
  
  //@ ghost seq<int> weight_seq  ;
  //@ assume |weight_seq| == A && (\forall int i; i >= 0 && i < A; weight_seq[i] == host_weight[i]);

  

  

  //@ loop_invariant V == 1024 && A == 10 * V;
  //@ loop_invariant source >= 0 && source < V;
  //@ loop_invariant counter >= 0 && counter <= V - 1;
  //@ loop_invariant |start_seq| == A;
  //@ loop_invariant |weight_seq| == A;
  //@ loop_invariant |end_seq| == A;
  //@ loop_invariant |cost_seq| == V;
  //@ loop_invariant |oldcost_seq| == V;
  //@ loop_invariant |contrib| == A;
  //@ loop_invariant |contrib2| == A;
  //@ loop_invariant (\forall int i; 0 <= i && i < V; oldcost_seq[i] == cost_seq[i]);
  //@ loop_invariant (\forall int i; 0 <= i && i < A; contrib[i] == 0);
  //@ loop_invariant (\forall int i; 0 <= i && i < A; contrib2[i] == 0);
  //@ loop_invariant (\forall* int i; 0 <= i && i < A; Perm(host_start[i], read)); 
  //@ loop_invariant (\forall int i; 0 <= i && i < A; host_start[i] >= 0 && host_start[i] < V);
  //@ loop_invariant (\forall* int i; 0 <= i && i < A; Perm(host_end[i], read)); 
  //@ loop_invariant (\forall int i; 0 <= i && i < A; host_end[i] >= 0 && host_end[i] < V);
  //@ loop_invariant (\forall int i; 0 <= i && i < A; host_start[i] != host_end[i]); 
  //@ loop_invariant (\forall int i; 0 <= i && i < A; (\forall int j; 0 <= j && j < A && i != j; host_start[i] == host_start[j] ==> host_end[i] != host_end[j]));		
  //@ loop_invariant (\forall int i; 0 <= i && i < A; host_start[i] == start_seq[i]);
  //@ loop_invariant (\forall int i; 0 <= i && i < A; host_end[i] == end_seq[i]);	
  //@ loop_invariant (\forall* int i; 0 <= i && i < A; Perm(host_weight[i], read));
  //@ loop_invariant (\forall int i; 0 <= i && i < A; host_weight[i] == weight_seq[i]); 
  //@ loop_invariant (\forall int i; 0 <= i && i < A; host_weight[i] > 0);							
  //@ loop_invariant (\forall* int i; 0 <= i && i < V; Perm(host_cost[i], write));
  //@ loop_invariant (\forall int i; 0 <= i && i < V && i != source; cost_seq[i] == inf() || cost_seq[i] > 0);
  //@ loop_invariant host_cost[source] == 0;  								
  //@ loop_invariant (\forall int i; 0 <= i && i < V; host_cost[i] == cost_seq[i]);
  //@ loop_invariant (\forall int i; 0 <= i && i < V && cost_seq[i] != inf(); ExPathEqual(V, A, start_seq, end_seq, weight_seq, source, i, cost_seq[i])); 
  while(counter < V-1)
  {  
    
    
    
    //@ ghost contrib = contrib2 ;
    //@ assert (\forall int i; i >= 0 && i < A; contrib[i] == 0);
    
    //@ ghost oldcost_seq = cost_seq ;
    //@ assert (\forall int i; i >= 0 && i < V; oldcost_seq[i] == cost_seq[i]);

    counter = counter + 1;
  }
  
  
  // copy result from device to host
  vercorsCudaMemcpyInt(host_cost, device_cost, V, hipMemcpyDeviceToHost);	
	 
  
  // cleanup memory
  vercorsFreeInt(host_start);
  vercorsFreeInt(host_end);
  vercorsFreeInt(host_weight);
  vercorsFreeInt(host_cost);
  vercorsCudaFreeInt(device_start);
  vercorsCudaFreeInt(device_end);
  vercorsCudaFreeInt(device_weight);
  vercorsCudaFreeInt(device_cost);

	return 0;
       
}

