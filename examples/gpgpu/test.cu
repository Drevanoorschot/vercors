#include "hip/hip_runtime.h"
/***********************************************************************************
Created by Mohsen Safari.
************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
//Kernel
////////////////////////////////////////////////////////////////////////////////

/*@
  yields seq<int> contrib; 
  given seq<int> cost_seq;
  given seq<int> oldcost_seq;
  given seq<int> start_seq;
  given seq<int> end_seq;
  given seq<int> weight_seq; 

  context_everywhere V == 1024 && A == 10 * V;
  context_everywhere opencl_gsize == V;
  context_everywhere 10 == opencl_gcount;
  context_everywhere source >= 0 && source < V;
  context_everywhere counter >= 0 && counter < V-1;
  context_everywhere |start_seq| == A && |end_seq| == A && |weight_seq| == A && |cost_seq| == V && |contrib| == A;
  
  kernel_invariant \pointer(g_start, A, 1\2);
  kernel_invariant \pointer(g_end, A, 1\2);
  kernel_invariant (\forall int i; 0 <= i && i < A; g_start[i] >= 0 && g_start[i] < V && g_end[i] >= 0 && g_end[i] < V);
  kernel_invariant (\forall int i; 0 <= i && i < A; g_start[i] != g_end[i]);
  kernel_invariant (\forall int i; 0 <= i && i < A; (\forall int j; 0 <= j && j < A && i != j; g_start[i] == g_start[j] ==> g_end[i] != g_end[j]));
  
  kernel_invariant \pointer(g_weight, A, 1\2);
  kernel_invariant (\forall int i; 0 <= i && i < A; g_weight[i] > 0);
  
  kernel_invariant \pointer(g_cost, V, write);

  //kernel_invariant (\forall int i; 0 <= i && i < A; g_start[i] == start_seq[i]);
  //kernel_invariant (\forall int i; 0 <= i && i < A; g_end[i] == end_seq[i]);
  //kernel_invariant (\forall int i; 0 <= i && i < A; g_weight[i] == weight_seq[i]); 
   
  

@*/
__global__ void CUDAKernel(int* g_start, int* g_end, int* g_weight, int* g_cost, int V, int A, int counter, int source)
{
  int tid = blockIdx.x * V + threadIdx.x;
  //@ assert tid == \gtid;
  atomicRelax(g_cost+g_end[tid], g_weight[tid], g_cost[g_start[tid]])
  
}
       
